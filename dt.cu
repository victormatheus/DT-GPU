#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <float.h>

#define BLOCK_SIZE 256

__global__ void euclidian_distance_transform_kernel(
  const unsigned char* img, float* dist, int w, int h)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const int N = w*h;
  
  if (i >= N)
  {   
    return; 
  }
  
  int cx = i % w;
  int cy = i / w;
  
  float minv = INFINITY;
  
  if (img[i] > 0)
  {
    minv = 0.0f;
  }
  else
  {
    for (int j = 0; j < N; j++)  
    {
        if (img[j] > 0)
        {
          int x = j % w;
          int y = j / w;
          float d = sqrtf( powf(float(x-cx), 2.0f) + powf(float(y-cy), 2.0f) );
          if (d < minv) minv = d;
        }
    }
  }

  dist[i] = minv;
}

void euclidian_distance_transform(unsigned char* img, float* dist, int w, int h) {

    hipError_t err;
    unsigned char *d_img;
    hipMalloc((void**) &d_img, w*h*sizeof(unsigned char));    
    hipMemcpy(d_img, img, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
    }
    
    float* d_dist;
    hipMalloc((void**) &d_dist, w*h*sizeof(float));
    //hipMemset(d_dist, 0, w*h*sizeof(float));
    
    dim3 block (BLOCK_SIZE,1,1);
    
    int gx = (w*h+BLOCK_SIZE-1)/BLOCK_SIZE;    
    dim3 grid(gx,1);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
    }

    euclidian_distance_transform_kernel <<<grid, block>>> (d_img, d_dist, w, h);
    hipDeviceSynchronize();
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
    }
    
    hipMemcpy(dist, d_dist, w*h*sizeof(float), hipMemcpyDeviceToHost);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
    }
    
    hipFree(d_img);
    hipFree(d_dist);
}



int main()
{
    char line[256];
    int w,h;
    int i;
    int v;
    
    FILE* f = fopen("img.pgm", "r");
    fgets(line, sizeof(line), f);
    fgets(line, sizeof(line), f);
    fgets(line, sizeof(line), f);
    sscanf(line, "%d %d", &w, &h);
    fgets(line, sizeof(line), f);
    
    printf("%d %d\n", w, h);
    
    unsigned char* img = (unsigned char*)malloc(sizeof(unsigned char)*w*h);
    float* dist = (float*)malloc(sizeof(float)*w*h);
    
    for (i=0; i<w*h; i++)
    {
        fgets(line, sizeof(line), f);
        sscanf(line, "%d", &v);
        img[i] = (v > 0)? 255 : 0;
        //if (img[i]==255) printf("wp: %d %d\n", i%w, i/w);
    }
    
    fclose(f);
    
    printf("start\n");
    hipEvent_t start,stop;
    float time=0;
    hipEventCreate(&start);
    hipEventCreate(&stop);      
    hipEventRecord(start,0);
    
    euclidian_distance_transform(img, dist, w, h);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time,start,stop);
    
    printf("end\n");
    printf("time: %f\n", time);
    
    FILE* f2 = fopen("output.pgm", "w");
    fprintf(f2, "P2\n");
    fprintf(f2, "#\n");
    fprintf(f2, "%d %d\n", w, h);
    fprintf(f2, "255\n");
    
    float max = 0.0f;
    for (i=0; i<w*h; i++)
    {
       max = (dist[i] > max)? dist[i] : max;
    }
    printf("max: %f\n", max);
    
    for (i=0; i<w*h; i++)
    {
        fprintf(f2, "%d\n", ((int)floor((255.0f*dist[i])/max)));
    }
    
    fclose(f2);
       
    free(img);
    free(dist);
    
    return 0;
}
